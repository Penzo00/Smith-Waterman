#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include "smith-waterman.h"

#define S_LEN 512  // Length of sequences
#define N 1000     // Number of sequences
#define MATCH 1    // Score for a match
#define MISMATCH -1 // Score for a mismatch
#define INS -2     // Score for an insertion
#define DEL -2     // Score for a deletion

// Function to get the current time (implementation not shown here)

// Main function
int main() {
    srand(time(NULL));  // Seed the random number generator
    char alphabet[5] = {'A', 'C', 'G', 'T', 'N'};  // DNA alphabet

    char *query, *reference;
    int *res;

    // Allocate pinned host memory
    hipHostMalloc(&query, N * S_LEN * sizeof(char));
    hipHostMalloc(&reference, N * S_LEN * sizeof(char));
    hipHostMalloc(&res, N * sizeof(int));

    // Initialize sequences with random characters
    for (int i = 0; i < N * S_LEN; i++) {
        query[i] = alphabet[rand() % 5];
        reference[i] = alphabet[rand() % 5];
    }

    char *d_query, *d_reference;
    int *d_res;

    // Allocate device memory
    hipMalloc(&d_query, N * S_LEN * sizeof(char));
    hipMalloc(&d_reference, N * S_LEN * sizeof(char));
    hipMalloc(&d_res, N * sizeof(int));

    // Copy sequences to device memory
    hipMemcpyAsync(d_query, query, N * S_LEN * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_reference, reference, N * S_LEN * sizeof(char), hipMemcpyHostToDevice);

    // Initialize cuDNN
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // Create tensor descriptors
    hipdnnTensorDescriptor_t query_desc, reference_desc, res_desc;
    hipdnnCreateTensorDescriptor(&query_desc);
    hipdnnCreateTensorDescriptor(&reference_desc);
    hipdnnCreateTensorDescriptor(&res_desc);

    // Set tensor descriptors
    hipdnnSetTensor4dDescriptor(query_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, 1, S_LEN, 1);
    hipdnnSetTensor4dDescriptor(reference_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, 1, S_LEN, 1);
    hipdnnSetTensor4dDescriptor(res_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, 1, S_LEN, 1);

    // Create convolution descriptor
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnCreateFilterDescriptor(&filter_desc);
    hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, N, 1, S_LEN, 1);

    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnCreateConvolutionDescriptor(&conv_desc);
    hipdnnSetConvolution2dDescriptor(conv_desc, 0, 0, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

    // Get convolution algorithm
    hipdnnConvolutionFwdAlgoPerf_t algo_perf;
    int returned_algo_count;
    hipdnnFindConvolutionForwardAlgorithm(cudnn, query_desc, filter_desc, conv_desc, res_desc, 1, &returned_algo_count, &algo_perf);
    hipdnnConvolutionFwdAlgo_t algo = algo_perf.algo;

    // Get workspace size
    size_t workspace_size;
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn, query_desc, filter_desc, conv_desc, res_desc, algo, &workspace_size);

    // Allocate workspace
    void *workspace;
    hipMalloc(&workspace, workspace_size);

    // Perform convolution
    float alpha = 1.0f, beta = 0.0f;
    hipdnnConvolutionForward(cudnn, &alpha, query_desc, d_query, filter_desc, d_reference, conv_desc, algo, workspace, workspace_size, &beta, res_desc, d_res);

    // Measure execution time
    double start_gpu = get_time();
    hipDeviceSynchronize();
    double end_gpu = get_time();

    // Copy results back to host
    hipMemcpyAsync(res, d_res, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("SW Time GPU with cuDNN: %.10lf\n", end_gpu - start_gpu);

    // Free device memory
    hipFree(d_query);
    hipFree(d_reference);
    hipFree(d_res);
    hipFree(workspace);

    // Free host memory
    hipHostFree(query);
    hipHostFree(reference);
    hipHostFree(res);

    // Destroy cuDNN resources
    hipdnnDestroyTensorDescriptor(query_desc);
    hipdnnDestroyTensorDescriptor(reference_desc);
    hipdnnDestroyTensorDescriptor(res_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroy(cudnn);

    return 0;
}